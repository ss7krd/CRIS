#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include<stdio>
#include<iostream>
#include<math.h>
#include<inferenceFiles.h>
using namespace std;

vector<double>applicationsSLO;
map<int,vector<double>>applicationsBatchSizeLatency;
map<int,int>applicationBatchSizeMapping;
map<int,double>applicationGPUSpaceNecessity;
map<int,double>applicationGPUFractionMapping;
map<int,vector<int>>applicationEachModelAccuracyMapping[10];
map<int,vector<int>>applicationEachModelPerBatchLatency[10];
map<int,int>applicationEachModelHowMuchRetrainTime[10];
map<int,int>applicationEachModelBestExitModel[10];
map<int,int>applicationEachModelDegreeOfImpact[10];
vector<double>howManyRequests;
map<int,int>applicationHowManyModelsMapping;
map<int,int>applicationModelRetrainingRequired[10];
total_gpu_space_necessity = 0;

__global__
void applicationRunning(hipCtx_t *context, int id, int totalNumThreadsThisContext){
	if (id == 0){
		inferenceServing(&context, id, applicationBatchSizeMapping[id], totalNumThreadsThisContext);
		retraining(&context, id, applicationBatchSizeMapping[id], totalNumThreadsThisContext);
	}
}
__global__
void inferenceServing(hipCtx_t *context, int id, int batchSize, int totalNumThreadsThisContext){
	const int streamNo = batchSize;
	int numThreadsPerStream = int(floor(totalNumThreadsThisContext/streamNo));
	hipStream_t streamList[streamNo];
	int blockSize = 128;
	for (int i = 0; i < streamNo; i++){
		hipStreamCreate(&streamList[i]);
		inferenceServingPerStream<<<blockSize, numThreadsPerStream>>>(&context, &streamList[i], id, i);
	}
}
__global__
void retraining(hipCtx_t *context, int id, int batchSize, int totalNumThreadsThisContext){
	const int streamNo = batchSize;
	int numThreadsPerStream = int(floor(totalNumThreadsThisContext/streamNo));
	hipStream_t streamList[streamNo];
	int blockSize = 128;
	for (int i = 0; i < streamNo; i++){
		hipStreamCreate(&streamList[i]);
		retrainingPerStream<<<blockSize, numThreadsPerStream>>>(&context, &streamList[i], id, i);
	}
}
int main(){
	for (int i = 0; i < applicationsSLO.size(); i++){
		currApplicationBatchSizeLatency = applicationsBatchSizeLatency[i]
		currMinimum = 10000000000
		currBatchSize = -1
		for (int j = 0; j < currApplicationBatchSizeLatency.size(); j++){
			if(currApplicationBatchSizeLatency[j] < currMinimum){
				currMinimum = currApplicationBatchSizeLatency[j]
				currBatchSize = j
			}
		}
		applicationBatchSizeMapping[i] = currBatchSize
		gpu_space_necessity = currApplicationBatchSizeLatency[currBatchSize]/applicationsSLO[i]
		applicationGPUSpaceNecessity[i] = gpu_space_necessity;
		total_gpu_space_necessity = total_gpu_space_necessity + gpu_space_necessity
	}
	for (int i = 0; i < applicationsSLO.size(); i++){
		applicationGPUFractionMapping[i] = applicationGPUSpaceNecessity[i]/total_gpu_space_necessity
	}

	for (int i = 0; i < applicationsSLO.size(); i++){
		currApplMap = applicationEachModelAccuracyMapping[i];
		for (map<int,vector<int>>::iterator itr=currApplMap.begin(); itr != currApplMap.end(); itr++){
			currModelId = itr->first;
			currExitModelsVector = itr->second;
			currMax = -100000
			currMaxId = -1
			for (int j = 0; j < currExitModelsVector.size(); j++){
				if (currExitModelsVector[j] > currMax){
					currMax = currExitModelsVector[j];
					currMaxId = j;
				}

			}
			applicationEachModelBestExitModel[i][currModelId] = currMaxId;
		}
	}

	vector<int>GPUSpaceAdjustFrom;
	vector<double>GPUSpaceAdjustFromTotalInfTime;
	vector<int>GPUSpaceAdjustTo;
	map<int,double>applicationTotalDegreeOfImpactMapping;
	for (int i = 0; i < applicationsSLO.size(); i++){
		int oneModelRetrain = 0;
		int gpuSpaceNeeded = 0;
		howManyBatches = howManyRequests[i]/applicationBatchSizeMapping[i];
		totalInferenceTime = 0;
		howManyModels = applicationHowManyModelsMapping[i];
		for (int j = 1; j <= howManyModels; j++){
			whatVersion = applicationEachModelBestExitModel[i][j];
			timePerBatch = applicationEachModelPerBatchLatency[i][j][whatVersion];
			totalTime = howManyBatches*timePerBatch;
			totalInferenceTime = totalInferenceTime + totalTime;
		}
		totalRetrainTimeRemaining = applicationsSLO[i] - totalInferenceTime;
		totalDegreeOfImpact = 0;
		for (int j = 1; j <= howManyModels; j++){
			totalDegreeOfImpact += applicationEachModelDegreeOfImpact[i][j];

		}
		applicationTotalDegreeOfImpactMapping[i] = totalDegreeOfImpact;
		for (int j = 1; j <= howManyModels; j++){
			if (applicationModelRetrainingRequired[i][j] == 1){
				oneModelRetrain = 1;
				applicationEachModelHowMuchRetrainTime[i][j] = totalRetrainTimeRemaining*(applicationEachModelDegreeOfImpact[i][j]/totalDegreeOfImpact);
				if (applicationEachModelHowMuchRetrainTime[i][j] == 0){
					gpuSpaceNeeded = 1;
				}
			}
		}
		if (oneModelRetrain == 0){
			GPUSpaceAdjustFrom.push_back(i);
			GPUSpaceAdjustFromTotalInfTime.push_back(totalInferenceTime);
		}
		if (gpuSpaceNeeded == 1){
			GPUSpaceAdjustTo.push_back(i);
		}
	}
	double totalExtraGPUFraction = 0;
	for (int i = 0; i < GPUSpaceAdjustFrom.size(); i++){
		currAppl = GPUAdjustFrom[i];
		totalExtraGPUFraction += applicationGPUFractionMapping[i]*(GPUSpaceAdjustFromTotalInfTime[i]/applicationsSLO[i]);
		applicationGPUFractionMapping[i] -= applicationGPUFractionMapping[i]*(GPUSpaceAdjustFromTotalInfTime[i]/applicationsSLO[i]);
	}
	degreeOfImpactSummation = 0;
	for (int i = 0; i < GPUSpaceAdjustTo.size(); i++){
		currAppl = GPUSpaceAdjustTo[i];
		degreeOfImpactSummation += applicationTotalDegreeOfImpactMapping[currAppl];
	}
	for (int i = 0; i < GPUSpaceAdjustTo.size(); i++){
		applicationGPUFractionMapping[i] = totalExtraGPUFraction*(applicationTotalDegreeOfImpactMapping[currAppl]/degreeOfImpactSummation);
	}



	for (int i = 0; i < applicationsSLO.size(); i++){
		currApplMap = applicationEachModelAccuracyMapping[i];
		for (map<int,vector<int>>::iterator itr=currApplMap.begin(); itr != currApplMap.end(); itr++){
			currModelId = itr->first;
			currExitModelsVector = itr->second;
			currMax = -100000
			currMaxId = -1
			for (int j = 0; j < currExitModelsVector.size(); j++){
				if (currExitModelsVector[j] > currMax){
					currMax = currExitModelsVector[j];
					currMaxId = j;
				}

			}
			applicationEachModelBestExitModel[i][currModelId] = currMaxId;
		}
	}

	for (int i = 0; i < applicationsSLO.size(); i++){
		int oneModelRetrain = 0;
		int gpuSpaceNeeded = 0;
		howManyBatches = howManyRequests[i]/applicationBatchSizeMapping[i];
		totalInferenceTime = 0;
		howManyModels = applicationHowManyModelsMapping[i];
		for (int j = 1; j <= howManyModels; j++){
			whatVersion = applicationEachModelBestExitModel[i][j];
			timePerBatch = applicationEachModelPerBatchLatency[i][j][whatVersion];
			totalTime = howManyBatches*timePerBatch;
			totalInferenceTime = totalInferenceTime + totalTime;
		}
		totalRetrainTimeRemaining = applicationsSLO[i] - totalInferenceTime;
		totalDegreeOfImpact = 0;
		for (int j = 1; j <= howManyModels; j++){
			totalDegreeOfImpact += applicationEachModelDegreeOfImpact[i][j];
		}
		for (int j = 1; j <= howManyModels; j++){
			if (applicationModelRetrainingRequired[i][j] == 1){
				applicationEachModelHowMuchRetrainTime[i][j] = totalRetrainTimeRemaining*(applicationEachModelDegreeOfImpact[i][j]/totalDegreeOfImpact);
			}
		}
	}

	const int CONTEXT_POOL_SIZE = applicationsSLO.size();
	int totalNumThreads = 163840;
	int blockSize = 256;
	hipCtx_t contextPool[CONTEXT_POOL_SIZE];
	for (int i = 0; i < applicationsSLO.size(); i++){
		cudaContextCreate(&contextPool[i]);
		cuda_mps_active_thread_percentage(&contextPool[i], applicationGPUFractionMapping[i]*100.0);
		double howManyThreads = floor(totalNumThreads * applicationGPUFractionMapping[i]);
		int threadsPerBlock = int(floor(howManyThreads/blockSize));
		applicationRunning<<<blockSize,threadsPerBlock>>>(&contextPool[i], i, blockSize*threadsPerBlock);
	}


	return 0;
}